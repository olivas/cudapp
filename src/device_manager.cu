#include <cudapp/device_manager.hpp>
#include <cudapp/check_error.cuh>

DeviceManager::DeviceManager()
{
  CHECK_ERROR(hipSetDeviceFlags(hipDeviceScheduleBlockingSync));
  CHECK_ERROR(hipSetDevice(device_number_));  
  std::cerr<<"device_number_ = "<<device_number_<<std::endl;

  hipDeviceProp_t device_properties;
  CHECK_ERROR(hipGetDeviceProperties(&device_properties, device_number_));
  total_global_mem_ = device_properties.totalGlobalMem;
  multi_processor_count_ = device_properties.multiProcessorCount;  
}

DeviceManager::~DeviceManager()
{  
  CHECK_ERROR(hipDeviceReset());
}

