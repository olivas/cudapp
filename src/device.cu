#include <cudapp/device.hpp>
#include <cudapp/check_error.cuh>

cudapp::Device::Device(int device_number):
  device_number_(device_number)
{
  CHECK_ERROR(hipSetDeviceFlags(hipDeviceScheduleBlockingSync));
  CHECK_ERROR(hipSetDevice(device_number_));  
  std::cerr<<"device_number_ = "<<device_number_<<std::endl;

  hipDeviceProp_t device_properties;
  CHECK_ERROR(hipGetDeviceProperties(&device_properties, device_number_));
  total_global_mem_ = device_properties.totalGlobalMem;
  multi_processor_count_ = device_properties.multiProcessorCount;
}

cudapp::Device::~Device()
{  
  CHECK_ERROR(hipDeviceReset());
}

