#include "hip/hip_runtime.h"
//
// Created by olivas on 4/15/24.
//
#include <stdlib.h>
#include <stdio.h>

#include <unordered_map>
#include <vector>
#include <cassert>

#include <hip/hip_runtime.h>
#include <cutensornet.h>


#define HANDLE_ERROR(x)                                           \
{ const auto err = x;                                             \
  if( err != CUTENSORNET_STATUS_SUCCESS )                         \
  { printf("Error: %s in line %d\n", cutensornetGetErrorString(err), __LINE__); \
    fflush(stdout);                                               \
  }                                                               \
};

#define HANDLE_CUDA_ERROR(x)                                      \
{ const auto err = x;                                             \
  if( err != hipSuccess )                                        \
  { printf("CUDA Error: %s in line %d\n", hipGetErrorString(err), __LINE__); \
    fflush(stdout);                                               \
  }                                                               \
};


struct GPUTimer
{
    GPUTimer(hipStream_t stream): stream_(stream)
    {
        HANDLE_CUDA_ERROR(hipEventCreate(&start_));
        HANDLE_CUDA_ERROR(hipEventCreate(&stop_));
    }

    ~GPUTimer()
    {
        HANDLE_CUDA_ERROR(hipEventDestroy(start_));
        HANDLE_CUDA_ERROR(hipEventDestroy(stop_));
    }

    void start()
    {
        HANDLE_CUDA_ERROR(hipEventRecord(start_, stream_));
    }

    float seconds()
    {
        HANDLE_CUDA_ERROR(hipEventRecord(stop_, stream_));
        HANDLE_CUDA_ERROR(hipEventSynchronize(stop_));
        float time;
        HANDLE_CUDA_ERROR(hipEventElapsedTime(&time, start_, stop_));
        return time * 1e-3;
    }

private:
    hipEvent_t start_, stop_;
    hipStream_t stream_;
};


int main() {
    static_assert(sizeof(size_t) == sizeof(int64_t), "Please build this sample on a 64-bit architecture!");

    bool verbose = true;

    // Check cuTensorNet version
    const size_t cuTensornetVersion = cutensornetGetVersion();
    if (verbose)
        printf("cuTensorNet version: %ld\n", cuTensornetVersion);

    // Set GPU device
    int numDevices{0};
    HANDLE_CUDA_ERROR(hipGetDeviceCount(&numDevices));
    const int deviceId = 0;
    HANDLE_CUDA_ERROR(hipSetDevice(deviceId));
    hipDeviceProp_t prop;
    HANDLE_CUDA_ERROR(hipGetDeviceProperties(&prop, deviceId));

    if (verbose) {
        printf("===== device info ======\n");
        printf("GPU-name:%s\n", prop.name);
        printf("GPU-clock:%d\n", prop.clockRate);
        printf("GPU-memoryClock:%d\n", prop.memoryClockRate);
        printf("GPU-nSM:%d\n", prop.multiProcessorCount);
        printf("GPU-major:%d\n", prop.major);
        printf("GPU-minor:%d\n", prop.minor);
        printf("========================\n");
    }

    typedef float floatType;
    hipDataType typeData = HIP_R_32F;
    cutensornetComputeType_t typeCompute = CUTENSORNET_COMPUTE_32F;

    if (verbose)
        printf("Included headers and defined data types\n");
}